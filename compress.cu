#include <math.h>
#include <iostream>
#include <array>
#include <cmath>
#include <cstdint>
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime_api.h>


using namespace std;

template<int E, int M, int T, int P, int B = (1 << (E - 1)) - 1>
static inline __device__ uint64_t compress(float* xyz)
{
    static_assert(E + M + P + T == 64, "Invalid number of bits");
    
    const float rpis = 1.0/static_cast<float>(M_PI);
    const double told = 1e-13;

    float ts = atan2(static_cast<float>(xyz[1]),static_cast<float>(xyz[0]));
    uint64_t nt = round(static_cast<float>((1 << (T-1)) - 1)*(abs(ts)*rpis));
    if(ts >= 0) nt = nt | (1 << (T-1));

    double rd = sqrt((double)xyz[0]*xyz[0] + (double)xyz[1]*xyz[1] + (double)xyz[2]*xyz[2]);
    float rs = (float)rd;
    float pd = (rd < told) ? 0 : static_cast<float>(acos(static_cast<double>(xyz[2])/rd));
    uint64_t np = round(static_cast<float>((1 << P)-1)*pd*rpis);

    uint32_t* ptr = reinterpret_cast<uint32_t*>(&rs);
    uint64_t fre = (*ptr & 0x7f800000) >> 23;
    uint64_t frm = (*ptr & 0x007fffff) >> 23-M ;
    
    return nt | (np << (T)) | (frm << (T + P)) | ((fre - 127 + B) << (T + P + M));
    
}

template<int E, int M, int T, int P, int B = (1 << (E - 1)) - 1>
static inline __device__ void decompress(const uint64_t& C, float& x, float& y, float& z)
{
    static_assert(E + M + P + T == 64, "Invalid number of bits");
    
    const float pi = static_cast<float>(M_PI);
    
    uint32_t tm = (1 << T) - 1;
    uint32_t pm = (1 << P) - 1;
    uint32_t mm = (1 << M) - 1;
    uint32_t em = (1 << E) - 1;
    
    uint32_t td = C & tm;
    uint32_t pd = (C >> T) & pm;
    uint32_t frm = (C >> (T+P)) & mm;
    uint32_t fre = (C >> (T+P+M)) & em;
    uint32_t ri = ((fre - B + 127) << 23) | (frm << (23-M));
    
    float r = *reinterpret_cast<float*>(&ri);    
    float t = pi*(-1.0 + 2.0*(td >> (T-1)))*(td & (tm >> 1))/static_cast<float>((1 << (T-1))-1);    
    float p = pi*pd/static_cast<float>((1 << P)-1);
    
    x = r*cosf(t)*sinf(p);
    y = r*sinf(t)*sinf(p);
    z = r*cosf(p);
    
}
